#include	<hip/hip_runtime.h>
#include	<stdio.h>
#include	<stdlib.h>
#include	"processor_cuda.h"

// #ifdef __cplusplus
// extern "C" {
// #endif

cuStreamPool* stream_pool_init()
{
	cuStreamPool* pool = (cuStreamPool*)malloc(sizeof(cuStreamPool));
	if (!pool)
	{
		return	NULL;
	}

	for (int i = 0; i < MAX_STREAMS; i++)
	{
		pool->streams[i] = NULL;
		pool->status[i] = stream_status_none;
	}
	pool->n_ref = 0;

	return	pool;
}

void stream_pool_destroy(cuStreamPool* pool)
{
	if (pool)
	{
		for (int i = 0; i < MAX_STREAMS; i++)
		{
			if (pool->streams[i] != NULL)
			{
				hipStreamDestroy((hipStream_t)pool->streams[i]);
				pool->streams[i] = NULL;
			}
		}
		free(pool);
	}
}

inline int	pick_stream(cuStreamPool* pool)
{
	int	i;

	for (i = 0; i < MAX_STREAMS && pool->status[i] != stream_status_available; i++);
	if (i >= MAX_STREAMS)
	{
		for (i = 0; i < MAX_STREAMS && pool->status[i] != stream_status_none; i++);
		if (i >= MAX_STREAMS)
			return	-1;
	}

	return	i;
}

int stream_pool_submit(cuStreamPool* pool, struct processor_cuda_t* p)
{
	if((p->istream = pick_stream(pool)) == -1)
	{
		return -1;
	}
	if (pool->status[p->istream] == stream_status_none)
		if (hipStreamCreate((hipStream_t*)(&pool->streams[p->istream])) != hipSuccess){
			ddq_warning("processor_cuda : Something is wrong when calling hipStreamCreate().\n");
			return -1;
		}
	pool->status[p->istream] = stream_status_inuse;
	
	p->ret = ((task_cuda_f *)(p->head.f->p))(p->head.p_inputs, p->head.p_outputs, p->head.p_attributes, pool->streams[p->istream]);

	return 1;
}

int stream_pool_query(cuStreamPool* pool, struct processor_cuda_t* p)
{
	if (hipStreamQuery((hipStream_t)pool->streams[p->istream]) != hipErrorNotReady)
	{
		pool->status[p->istream] = stream_status_available;
		return 1;
	}
	return 0;
}

// #ifdef __cplusplus
// }
// #endif