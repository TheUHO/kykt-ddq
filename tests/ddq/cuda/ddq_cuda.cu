#include "hip/hip_runtime.h"
#include	<stdio.h>
#include	<stdlib.h>
#include	<unistd.h>
#include <string.h>

#include	"ddq.h"
#include	"oplib.h"
#include    "error.h"

#include    "std/std_ops/std_ops.h"
#include    "task_types.h"
#include "dag.h"
#include "matrix.h"
#include "ddq_dag.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "task_types.h"

#include "ddq_cuda.h"

// #define SIZE 256
extern int row_size;
extern int col_size;
extern int reduce_size;

// 随机生成矩阵
void generate_random_matrix(double* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = (double)rand() / RAND_MAX; // 生成 [0, 1) 的随机数
    }
}

void* cuMatrix_new(){
    double* res;
    hipMalloc((void**)&res, row_size * reduce_size * sizeof(double));
    return res;
}
void cuMatrix_delete(void* p){
    hipFree(p);
}

task_ret op_cu_matmul_optimized(void** inputs, void** outputs, void** attributes, void* stream) {
    // 输入矩阵和输出矩阵
    
    double* imat0 = (double*)inputs[0]; // 输入矩阵 A
    double* imat1 = (double*)inputs[1]; // 输入矩阵 B
    double* omat = (double*)outputs[0]; // 输出矩阵 C

    double* d_imat0 = (double*)inputs[2];
    double* d_imat1 = (double*)inputs[3];
    double* d_omat = (double*)outputs[1];

    // 分配 GPU 内存
    // double *d_imat0, *d_imat1, *d_omat;
    // hipMalloc((void**)&d_imat0, row_size * reduce_size * sizeof(double));
    // hipMalloc((void**)&d_imat1, reduce_size * col_size * sizeof(double));
    // hipMalloc((void**)&d_omat, row_size * col_size * sizeof(double));

    // 将输入矩阵从 Host 拷贝到 Device
    hipMemcpyAsync(d_imat0, imat0,row_size * reduce_size * sizeof(double), hipMemcpyHostToDevice, (hipStream_t)stream);
    hipMemcpyAsync(d_imat1, imat1, reduce_size * col_size * sizeof(double), hipMemcpyHostToDevice, (hipStream_t)stream);
    hipMemcpyAsync(d_omat, omat, row_size * col_size * sizeof(double), hipMemcpyHostToDevice, (hipStream_t)stream);

    // 创建 cuBLAS 句柄并绑定流
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, (hipStream_t)stream);

    // 执行矩阵乘法 C = alpha * A * B + beta * C
    double alpha = 1.0;
    double beta = 1.0;
    hipblasDgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N, // 不转置 A 和 B
                col_size, row_size, reduce_size,                  // 矩阵维度
                &alpha,                   // alpha
                d_imat1, col_size,               // B 和其 leading dimension
                d_imat0, reduce_size,               // A 和其 leading dimension
                &beta,                    // beta
                d_omat, col_size);               // C 和其 leading dimension

    // 将结果从 Device 拷贝回 Host
    hipMemcpyAsync(omat, d_omat, row_size * col_size * sizeof(double), hipMemcpyDeviceToHost, (hipStream_t)stream);
    
    // 同步流，确保操作完成
    // hipStreamSynchronize((hipStream_t)stream);

    // 释放 GPU 内存和 cuBLAS 句柄
    // hipFree(d_imat0);
    // hipFree(d_imat1);
    // hipFree(d_omat);
    // hipblasDestroy(handle);
        // printf("op_cu_matmul_optimized: %p %p %p %p\n", imat0, imat1, omat, stream);
    return task_ret_ok;
}

ddq_ring matmul_ring(double* imat0, double* imat1, double* omat) {

    ddq_ring ring = ddq_new(NULL, 0, 0);
    
    obj f_matmul;
    f_matmul = obj_import(ring, (void*)op_cu_matmul_optimized, NULL, obj_prop_ready);
    
    obj obj_imat0, obj_imat1, obj_omat;
    obj_imat0 = obj_import(ring, imat0, NULL, obj_prop_ready | obj_prop_consumable);
    obj_imat1 = obj_import(ring, imat1, NULL, obj_prop_ready | obj_prop_consumable);
    obj_omat = obj_import(ring, omat, NULL, obj_prop_consumable);
    obj obj_d_imat0, obj_d_imat1, obj_d_omat;
    obj_d_imat0 = obj_new(ring, cuMatrix_new, cuMatrix_delete, obj_prop_consumable);
    obj_d_imat1 = obj_new(ring, cuMatrix_new, cuMatrix_delete, obj_prop_consumable);
    obj_d_omat = obj_new(ring, cuMatrix_new, cuMatrix_delete, obj_prop_consumable);

    ddq_op matmul = ddq_spawn(ring, processor_cuda, 4, 2);
    ddq_add_f(matmul, f_matmul);
    ddq_add_inputs(matmul, 0, obj_imat0);
    ddq_add_inputs(matmul, 1, obj_imat1);
    ddq_add_inputs(matmul, 2, obj_d_imat0);
    ddq_add_inputs(matmul, 3, obj_d_imat1);
    ddq_add_outputs(matmul, 0, obj_omat);
    ddq_add_outputs(matmul, 1, obj_d_omat);

    return ring;
}

// 根据 DAG 自动生成 op_cu_matmul_optimized 算子图
ddq_ring generate_ddq_from_dag_cuda(int num_nodes, const NodeInfo* nodes, double** matrices) {
    printf("生成基于 CUDA 的 DAG 算子图\n");

    // 创建一个新的 ddq_ring
    ddq_ring ring = ddq_new(NULL, 0, 0);

    // 导入 op_cu_matmul_optimized 算子
    obj matmul = obj_import(ring, (void*)op_cu_matmul_optimized, NULL, obj_prop_ready);

    // 创建 op 和输入输出对象
    ddq_op* ops = (ddq_op*)malloc(num_nodes * sizeof(ddq_op));
    int* inputs = (int*)malloc(num_nodes * sizeof(int));
    int nothing_inputs = 0;

    for (int i = 0; i < num_nodes; i++) {
        // 为每个节点创建一个 CUDA 处理器的 op
        ops[i] = ddq_spawn(ring, processor_cuda, 4 + nodes[i].input_count, 2);

        // 创建输入和输出对象
        obj imat0 = obj_import(ring, matrices[i * 3], NULL, obj_prop_ready | obj_prop_consumable);
        obj imat1 = obj_import(ring, matrices[i * 3 + 1], NULL, obj_prop_ready | obj_prop_consumable);
        obj omat = obj_import(ring, matrices[i * 3 + 2], NULL, obj_prop_consumable);

        // 创建 GPU 上的矩阵对象
        obj d_imat0 = obj_new(ring, cuMatrix_new, cuMatrix_delete, obj_prop_consumable);
        obj d_imat1 = obj_new(ring, cuMatrix_new, cuMatrix_delete, obj_prop_consumable);
        obj d_omat = obj_new(ring, cuMatrix_new, cuMatrix_delete, obj_prop_consumable);

        // 添加算子函数和输入输出
        ddq_add_f(ops[i], matmul);
        ddq_add_inputs(ops[i], 0, imat0);
        ddq_add_inputs(ops[i], 1, imat1);
        ddq_add_inputs(ops[i], 2, d_imat0);
        ddq_add_inputs(ops[i], 3, d_imat1);
        ddq_add_outputs(ops[i], 0, omat);
        ddq_add_outputs(ops[i], 1, d_omat);

        // inputs[i] = 4;

        // 如果节点没有输出，则计入 nothing_inputs
        // if (nodes[i].output_count == 0) {
        //     nothing_inputs++;
        // }
    }

    // 创建一个 "nothing" 算子，用于处理没有输出的节点
    // obj nothing = obj_import(ring, op_nothing, NULL, obj_prop_ready);
    // ddq_op nothing_op = ddq_spawn(ring, processor_cuda, nothing_inputs, 0);
    // ddq_add_f(nothing_op, nothing);

    // 连接算子之间的依赖关系
    for (int i = 0; i < num_nodes; i++) {
        // if (nodes[i].output_count == 0) {
        //     ddq_add_inputs(nothing_op, --nothing_inputs, ops[i]->outputs[0]);
        // }
        for (int j = 0; j < nodes[i].output_count; j++) {
            int to_node = nodes[i].outputs[j];
            ddq_add_inputs(ops[to_node], inputs[to_node]++, ops[i]->outputs[0]);
        }
    }

    // 释放临时分配的内存
    free(ops);
    free(inputs);

    return ring;
}

// int main(){
//     // 初始化随机数种子
//     srand(time(NULL));

//     double* imat0 = (double*)malloc(row_size * reduce_size * sizeof(double));
//     double* imat1 = (double*)malloc(reduce_size * col_size * sizeof(double));
//     double* omat = (double*)malloc(row_size * col_size * sizeof(double));
//     // 随机生成矩阵数据
//     generate_random_matrix(imat0, row_size, reduce_size);
//     generate_random_matrix(imat1, reduce_size, col_size);

//     // 初始化输出矩阵
//     for (int i = 0; i < row_size * col_size; i++) {
//         omat[i] = 0;
//     }
//     ddq_ring ring = matmul_ring(imat0, imat1, omat);
//     ddq_loop_init();
//     ddq_update(ring);
//     ddq_loop(ring, 0);
//     ddq_delete(ring);
    
//     //检查结果是否正确
//     double* check = (double*)malloc(row_size * col_size * sizeof(double));
//     for (int i = 0; i < row_size; i++) {
//         for (int j = 0; j < col_size; j++) {
//             check[i * col_size + j] = 0;
//             for (int k = 0; k < reduce_size; k++) {
//                 check[i * col_size + j] += imat0[i * reduce_size + k] * imat1[k * col_size + j];
//             }
//         }
//     }
//     for (int i = 0; i < row_size * col_size; i++) {
//         if (fabs(omat[i] - check[i]) > 1e-6) {
//             printf("Error: %f != %f\n", omat[i], check[i]);
//             break;
//         }
//     }
//     printf("Result is correct!\n");
//     free(check);
//     free(imat0);
//     free(imat1);
//     free(omat);
//     return 0;
// }

// int main() {
//     int num_nodes = 10;
//     int num_edges = 20;

//     // 生成随机 DAG
//     NodeInfo* nodes = generate_random_dag(num_nodes, num_edges);

//     // 分配矩阵数据
//     double** matrices = (double**)malloc(num_nodes * 3 * sizeof(double*));
//     for (int i = 0; i < num_nodes * 3; i++) {
//         matrices[i] = (double*)malloc(SIZE * SIZE * sizeof(double));
//         generate_random_matrix(matrices[i], SIZE, SIZE);
//     }

//     // 生成基于 CUDA 的算子图
//     ddq_ring ring = generate_ddq_from_dag_cuda(num_nodes, nodes, matrices);

//     // 执行任务流
//     ddq_loop_init();
//     ddq_update(ring);
//     ddq_loop(ring, 0);
//     ddq_delete(ring);

//     // 检查结果（可选）
//     // ...

//     // 释放内存
//     for (int i = 0; i < num_nodes * 3; i++) {
//         free(matrices[i]);
//     }
//     free(matrices);
//     free_node_info(nodes, num_nodes);

//     return 0;
// }