#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <chrono>
//nvcc -std=c++17 -O2 -o test test.cu -lcublas
// 随机生成矩阵
void generate_random_matrix(double* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = (double)rand() / RAND_MAX; // 生成 [0, 1) 的随机数
    }
}

// 执行单个 CUDA 节点的计算
void execute_cuda_node(double* h_A, double* h_B, double* h_C, int m, int n, int k, hipStream_t stream) {
    // 分配 GPU 内存
    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(double));
    hipMalloc((void**)&d_B, k * n * sizeof(double));
    hipMalloc((void**)&d_C, m * n * sizeof(double));

    // 将 Host 数据拷贝到 Device
    hipMemcpyAsync(d_A, h_A, m * k * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, k * n * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_C, h_C, m * n * sizeof(double), hipMemcpyHostToDevice, stream);

    // 创建 cuBLAS 句柄并绑定流
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, stream);

    // 设置矩阵乘法参数
    double alpha = 1.0;
    double beta = 1.0;

    // 执行矩阵乘法 C = alpha * A * B + beta * C
    hipblasDgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N, // 不转置 A 和 B
                n, m, k,                  // 矩阵维度
                &alpha,                   // alpha
                d_B, n,                   // B 和其 leading dimension
                d_A, k,                   // A 和其 leading dimension
                &beta,                    // beta
                d_C, n);                  // C 和其 leading dimension

    // 将结果从 Device 拷贝回 Host
    hipMemcpyAsync(h_C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost, stream);

    // 同步流，确保操作完成
    hipStreamSynchronize(stream);

    // 释放 GPU 内存和 cuBLAS 句柄
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}

// 执行完全并行的 CUDA 节点计算
void execute_parallel_nodes(int num_nodes, double** matrices, int m, int n, int k) {
    // 为每个节点分配一个 CUDA 流
    hipStream_t* streams = (hipStream_t*)malloc(num_nodes * sizeof(hipStream_t));
    for (int i = 0; i < num_nodes; i++) {
        hipStreamCreate(&streams[i]);
    }

    // 记录总执行时间
    auto start_time = std::chrono::high_resolution_clock::now();

    // 并行执行每个节点的计算
    for (int i = 0; i < num_nodes; i++) {
        // printf("执行节点 %d 的计算\n", i);

        // 获取当前节点的输入和输出矩阵
        double* h_A = matrices[i * 3];       // 输入矩阵 A
        double* h_B = matrices[i * 3 + 1];   // 输入矩阵 B
        double* h_C = matrices[i * 3 + 2];   // 输出矩阵 C

        // 执行 CUDA 节点计算
        execute_cuda_node(h_A, h_B, h_C, m, n, k, streams[i]);
    }

    // 等待所有流完成
    for (int i = 0; i < num_nodes; i++) {
        hipStreamSynchronize(streams[i]);
    }

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> total_elapsed_time = end_time - start_time;

    // 计算总性能（GFLOPS）
    double flops_per_task = 2.0 * m * n * k;
    double total_flops = num_nodes * flops_per_task;
    double gflops = total_flops / (total_elapsed_time.count() * 1e9);

    printf("总执行时间: %.6f 秒\n", total_elapsed_time.count());
    printf("总性能: %.2f GFLOPS\n", gflops);

    // 销毁 CUDA 流
    for (int i = 0; i < num_nodes; i++) {
        hipStreamDestroy(streams[i]);
    }
    free(streams);
}

int main() {
    // 初始化随机数种子
    srand(time(NULL));

    // 矩阵维度
    const int m = 4096; // A 的行数和 C 的行数
    const int n = 4096; // B 的列数和 C 的列数
    const int k = 4096; // A 的列数和 B 的行数

    // 节点数量
    int num_nodes = 10;

    // 分配矩阵数据
    double** matrices = (double**)malloc(num_nodes * 3 * sizeof(double*));
    for (int i = 0; i < num_nodes * 3; i++) {
        matrices[i] = (double*)malloc(m * n * sizeof(double));
        generate_random_matrix(matrices[i], m, n);
    }

    // 执行完全并行的节点计算
    execute_parallel_nodes(num_nodes, matrices, m, n, k);

    // 打印结果（仅用于调试）
    // for (int i = 0; i < num_nodes; i++) {
    //     char name[32];
    //     snprintf(name, sizeof(name), "节点 %d 的输出矩阵", i);
    //     print_matrix(name, matrices[i * 3 + 2], m, n);
    // }

    // 释放内存
    for (int i = 0; i < num_nodes * 3; i++) {
        free(matrices[i]);
    }
    free(matrices);

    return 0;
}